#include "mdl.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

/*
def lire_ligne_model(ligne):
	with open("/home/vadim/Bureau/Filtres-V1.4+ (versions)/3b1a/lignes_brute.bin", "rb") as co: bins = co.read()
	import struct as st
	import matplotlib.pyplot as plt
	BLOQUES, PRIXS = st.unpack('II', bins[:8])
	__lignes = [st.unpack('f'*PRIXS, bins[8+(i*PRIXS)*4:8+4*(i+1)*PRIXS]) for i in range(BLOQUES)]
	plt.plot(__lignes[ligne]);plt.show()
*/

char * nom_sources[SOURCES] = {
	" prixs ",
	"volumes",
	"  haut ",
	"  bas  "
};

//	Sources
float   prixs[PRIXS] = {};
float volumes[PRIXS] = {};
float   hight[PRIXS] = {};
float     low[PRIXS] = {};

float *          prixs__d = 0x0;
float *        volumes__d = 0x0;
float *          hight__d = 0x0;
float *            low__d = 0x0;

float * sources[SOURCES] = {
	prixs, volumes, hight, low
};

float * sources__d[SOURCES] = {
	prixs__d, volumes__d, hight__d, low__d
};

void charger_les_prixs() {
	uint __PRIXS;
	FILE * fp;
	//
	fp = fopen("prixs/prixs.bin", "rb");
	ASSERT(fp != 0);
	(void)!fread(&__PRIXS, sizeof(uint), 1, fp);
	ASSERT(__PRIXS == PRIXS);
	(void)!fread(prixs, sizeof(float), PRIXS, fp);
	fclose(fp);
	//
	fp = fopen("prixs/volumes.bin", "rb");
	ASSERT(fp != 0);
	(void)!fread(&__PRIXS, sizeof(uint), 1, fp);
	ASSERT(__PRIXS == PRIXS);
	(void)!fread(volumes, sizeof(float), PRIXS, fp);
	fclose(fp);
	//
	fp = fopen("prixs/hight.bin", "rb");
	ASSERT(fp != 0);
	(void)!fread(&__PRIXS, sizeof(uint), 1, fp);
	ASSERT(__PRIXS == PRIXS);
	(void)!fread(hight, sizeof(float), PRIXS, fp);
	fclose(fp);
	//
	fp = fopen("prixs/low.bin", "rb");
	ASSERT(fp != 0);
	(void)!fread(&__PRIXS, sizeof(uint), 1, fp);
	ASSERT(__PRIXS == PRIXS);
	(void)!fread(low, sizeof(float), PRIXS, fp);
	fclose(fp);
};

//	===========================================================

void ema_int_calc_ema(ema_int_t * ema_int) {
	//			-- Parametres --
	uint K = ema_int->K_ema;
	float _K = 1.0 / ((float)K);
	//	EMA
	ema_int->ema[0] = sources[ema_int->source][0];
	FOR(1, i, PRIXS) {
		ema_int->ema[i] = ema_int->ema[i-1] * (1.0 - _K) + sources[ema_int->source][i]*_K;
	}
};

//	===========================================================

nature_f fonctions_nature[NATURES] = {
	nature0__direct,
	nature1__macd,
	nature2__chiffre
};

uint NATURE_PARAMS[NATURES] = {
	0,
	1,
	1
};

uint min_param[NATURES][MAX_PARAMS] = {
	{0,0,0,0},
	{1,0,0,0},
	{1,0,0,0}
};

uint max_param[NATURES][MAX_PARAMS] = {
	{0,             0,       0,        0      }, 
	{MAX_COEF_MACD, 0,       0,        0      },
	{10000,         0,       0,        0      }
};

char * nom_natures[NATURES] {
	"directe",
	"  macd ",
	"chiffre",
};

ema_int_t * cree_ligne(uint source, uint nature, uint K_ema, uint intervalle, uint decale, uint params[MAX_PARAMS]) {
	ema_int_t * ret = alloc<ema_int_t>(1);
	//
	ret->source = source;
	ret->nature = nature;
	ret->K_ema  = K_ema;
	ret->intervalle = intervalle;
	ret->decale = decale;
	//
	ASSERT(intervalle <= MAX_INTERVALLE);
	ASSERT(decale     <= MAX_DECALES);
	ASSERT(K_ema      <= MAX_EMA);
	//
	memcpy(ret->params, params, sizeof(uint) * MAX_PARAMS);
	//
	ema_int_calc_ema(ret);
	fonctions_nature[nature](ret);
	//
	return ret;
};

void liberer_ligne(ema_int_t * ema_int) {

};

void charger_vram_nvidia() {
	CONTROLE_CUDA(hipMalloc((void**)&  prixs__d, sizeof(float) * PRIXS));
	CONTROLE_CUDA(hipMalloc((void**)&volumes__d, sizeof(float) * PRIXS));
	CONTROLE_CUDA(hipMalloc((void**)&  hight__d, sizeof(float) * PRIXS));
	CONTROLE_CUDA(hipMalloc((void**)&    low__d, sizeof(float) * PRIXS));
	//
	CONTROLE_CUDA(hipMemcpy(  prixs__d,   prixs, sizeof(float) * PRIXS, hipMemcpyHostToDevice));
	CONTROLE_CUDA(hipMemcpy(volumes__d, volumes, sizeof(float) * PRIXS, hipMemcpyHostToDevice));
	CONTROLE_CUDA(hipMemcpy(  hight__d, volumes, sizeof(float) * PRIXS, hipMemcpyHostToDevice));
	CONTROLE_CUDA(hipMemcpy(    low__d, volumes, sizeof(float) * PRIXS, hipMemcpyHostToDevice));
};

void     liberer_cudamalloc() {
	CONTROLE_CUDA(hipFree(  prixs__d));
	CONTROLE_CUDA(hipFree(volumes__d));
	CONTROLE_CUDA(hipFree(  hight__d));
	CONTROLE_CUDA(hipFree(    low__d));
};

void charger_tout() {
	printf("charger_les_prixs : ");    MESURER(charger_les_prixs());
	printf("charger_vram_nvidia : ");  MESURER(charger_vram_nvidia());
};

void liberer_tout() {
	titre("Liberer tout");
	liberer_cudamalloc();
};

ema_int_t * lire_ema_int(FILE * fp) {
	uint source, nature, K_ema, intervalle, decale;
	uint params[MAX_PARAMS];
	FREAD(&source,     sizeof(uint), 1, fp);
	FREAD(&nature,     sizeof(uint), 1, fp);
	FREAD(&K_ema,      sizeof(uint), 1, fp);
	FREAD(&intervalle, sizeof(uint), 1, fp);
	FREAD(&decale,     sizeof(uint), 1, fp);
	//
	FREAD(&params,     sizeof(uint), MAX_PARAMS, fp);
	//
	return cree_ligne(source, nature, K_ema, intervalle, decale, params);
};

void      ecrire_ema_int(ema_int_t * ema_int, FILE * fp) {
	FWRITE(&ema_int->source,     sizeof(uint), 1, fp);
	FWRITE(&ema_int->nature,     sizeof(uint), 1, fp);
	FWRITE(&ema_int->K_ema,      sizeof(uint), 1, fp);
	FWRITE(&ema_int->intervalle, sizeof(uint), 1, fp);
	FWRITE(&ema_int->decale,     sizeof(uint), 1, fp);
	//
	FWRITE(&ema_int->params,     sizeof(uint), MAX_PARAMS, fp);
};