#include "hip/hip_runtime.h"
#include "marchee.cuh"

void _outil_ema(float * y, float * x, uint K) {
	float _K = 1 / ((float)K);
	y[0] = x[0];
	FOR(1, t, PRIXS) {
		y[t] = y[t-1]*(1 - _K) + x[t] * _K;
	}
};

void _outil_macd(float * y, float * x, float coef) {
	ASSERT(coef > 0.0);
	float ema12[PRIXS], ema26[PRIXS], ema9[PRIXS], __macd[PRIXS];
	_outil_ema(ema12, x, 12*coef);
	_outil_ema(ema26, x, 26*coef);
	FOR(0, i, PRIXS) __macd[i] = ema12[i] - ema26[i];
	_outil_ema(ema9, __macd, 12*coef);
	FOR(0, i, PRIXS) y[i] = __macd[i] - ema9[i];
};

void _outil_chiffre(float * y, float * x, float chiffre) {
	FOR(0, t, PRIXS) {
		y[t] = 2*(chiffre-MIN2(fabs(x[t]-chiffre*roundf((x[t]+0)/chiffre)), fabs(x[t]-chiffre*roundf((x[t]+chiffre)/chiffre))))/chiffre-1;
	}
};