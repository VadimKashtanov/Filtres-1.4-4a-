#include "hip/hip_runtime.h"
#include "main.cuh"

/*
##	1) mdl norme theorique (borne) max et min pas du filtre, mais constant
##	2) mdl T bloque (pas *PRIXS) mdl_cree(T=16*16*1)
	3) plus de filtres
	4) filtres (d=1, decale=(0,4,8,16)), (d=32, decale=(0,8)), (d=256, decale=(0))
	5) dot1d_bloque 8172 -> 4*8172 -> 2*8172 -> 1*8172 -> 4096 -> 2048 -> 1024 -> 512 -> 256 -> 128 -> 64 -> 32 -> 16 -> 8 -> 4 -> 2 -> 1
	6) 256 -> 512 -> 512 -> 256 -> 512 -> 512 -> 256 -> ...
*/

#include "../impl_tmpl/tmpl_etc.cu"

static void plume_pred(Mdl_t * mdl, uint t0, uint t1) {
	uint fois = (t1-t0)/mdl->T;
	//
	float moyenne[P] = {0};
	//
	FOR(0, i, fois) {
		float * ancien = mdl_pred(mdl, t0 + i*mdl->T, (i+1)*mdl->T, 3);
		FOR(0, p, P) moyenne[p] += ancien[p];
		free(ancien);
	}
	printf("PRED GENERALE = ");
	FOR(0, p, P) printf(" %f%% ", 100*moyenne[p]/(float)fois);
	printf("\n");
};

float pourcent_masque_nulle[C] = {0};

/*float pourcent_masque[C] = {
	.10,
	.10,
	.10,
	.10,
	.10,
	.10,
	.10,
	.10,
	.10,
	.10,
	.00
};*/

float * alpha = de_a(1e-4, 1e-4, C);

uint optimiser_tous_les[C] = UNIFORME_C(1);

#define GRAND_T (16*16*1)

PAS_OPTIMISER()
int main(int argc, char ** argv) {
	alpha[0] = 1e-2;
	
	//	-- Init --
	srand(0);
	hipSetDevice(0);
	titre(" Charger tout ");   charger_tout();

	//	-- Verification --
	titre("Verifier MDL");     verif_mdl_1e5();

	//===============
	titre("  Programme Generale  ");
	ecrire_structure_generale("structure_generale.bin");

	/*uint Y[C] = {
		2048,
		1024,
		512,
		256,
		128,
		64,
		32,
		16,
		8,
		4,
		P
	};
	uint insts[C] = {
		FILTRES_PRIXS,
		DOT1D,
		DOT1D,
		DOT1D,
		DOT1D,
		DOT1D,
		DOT1D,
		DOT1D,
		DOT1D,
		DOT1D,
		DOT1D
	};
	//
	//	Assurances :
	ema_int_t * bloque[BLOQUES] = {
	//			    Source,      Nature,  K_ema, Intervalle, decale,     {params}
	// ----
		cree_ligne(SRC_PRIXS, DIRECT, 1, 1, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 1, 1, 8, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 1, 1, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 1, 1, 8, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 1, 8, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 1, 8, 8, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 4, 4, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 4, 4, 8, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 4, 4, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 4, 4, 8, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 4, 32, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 4, 32, 8, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 8, 1.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 8, 1.0, 8, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 8, 1.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 8, 1.0, 8, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 8, 8, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 8, 8, 8, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 8, 64, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 8, 64, 8, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 16, 2.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 16, 2.0, 8, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 16, 2.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 16, 2.0, 8, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 16, 16, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 16, 16, 8, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 16, 128, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 16, 128, 8, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 64, 8.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 64, 8.0, 8, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 64, 64, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 64, 64, 8, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 128, 16.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 128, 16.0, 8, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 128, 128, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 128, 128, 8, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 256, 32.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 256, 32.0, 8, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 256, 256, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 256, 256, 8, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 1, 1, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 1, 1, 8, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 1, 1, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 1, 1, 8, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 1, 8, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 1, 8, 8, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 4, 4, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 4, 4, 8, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 4, 4, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 4, 4, 8, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 4, 32, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 4, 32, 8, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 8, 1.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 8, 1.0, 8, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 8, 1.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 8, 1.0, 8, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 8, 8, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 8, 8, 8, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 8, 64, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 8, 64, 8, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 16, 2.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 16, 2.0, 8, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 16, 2.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 16, 2.0, 8, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 16, 16, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 16, 16, 8, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 16, 128, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 16, 128, 8, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 64, 8.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 64, 8.0, 8, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 64, 64, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 64, 64, 8, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 128, 16.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 128, 16.0, 8, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 128, 128, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 128, 128, 8, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 256, 32.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 256, 32.0, 8, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 256, 256, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 256, 256, 8, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 1, 1, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 1, 1, 8, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 1, 1, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 1, 1, 8, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 1, 8, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 1, 8, 8, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 4, 4, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 4, 4, 8, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 4, 4, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 4, 4, 8, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 4, 32, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 4, 32, 8, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 8, 1.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 8, 1.0, 8, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 8, 1.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 8, 1.0, 8, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 8, 8, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 8, 8, 8, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 8, 64, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 8, 64, 8, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 16, 2.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 16, 2.0, 8, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 16, 2.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 16, 2.0, 8, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 16, 16, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 16, 16, 8, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 16, 128, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 16, 128, 8, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 64, 8.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 64, 8.0, 8, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 64, 64, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 64, 64, 8, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 128, 16.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 128, 16.0, 8, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 128, 128, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 128, 128, 8, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 256, 32.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 256, 32.0, 8, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 256, 256, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 256, 256, 8, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 1, 1, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 1, 1, 8, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 1, 1, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 1, 1, 8, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 1, 8, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 1, 8, 8, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 4, 4, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 4, 4, 8, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 4, 4, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 4, 4, 8, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 4, 32, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 4, 32, 8, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 8, 1.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 8, 1.0, 8, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 8, 1.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 8, 1.0, 8, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 8, 8, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 8, 8, 8, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 8, 64, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 8, 64, 8, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 16, 2.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 16, 2.0, 8, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 16, 2.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 16, 2.0, 8, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 16, 16, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 16, 16, 8, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 16, 128, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 16, 128, 8, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 64, 8.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 64, 8.0, 8, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 64, 64, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 64, 64, 8, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 128, 16.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 128, 16.0, 8, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 128, 128, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 128, 128, 8, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 256, 32.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 256, 32.0, 8, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 256, 256, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 256, 256, 8, cree_DIRECTE()),
	// ----
		cree_ligne(SRC_PRIXS, MACD, 1, 1, 0, cree_MACD(1)),
		cree_ligne(SRC_PRIXS, MACD, 1, 1, 8, cree_MACD(1)),
		cree_ligne(SRC_PRIXS, MACD, 4, 4, 0, cree_MACD(1)),
		cree_ligne(SRC_PRIXS, MACD, 4, 4, 8, cree_MACD(1)),
		cree_ligne(SRC_PRIXS, MACD, 16, 1.0, 0, cree_MACD(1)),
		cree_ligne(SRC_PRIXS, MACD, 16, 1.0, 8, cree_MACD(1)),
		cree_ligne(SRC_PRIXS, MACD, 16, 16, 0, cree_MACD(1)),
		cree_ligne(SRC_PRIXS, MACD, 16, 16, 8, cree_MACD(1)),
		cree_ligne(SRC_PRIXS, MACD, 64, 4.0, 0, cree_MACD(1)),
		cree_ligne(SRC_PRIXS, MACD, 64, 4.0, 8, cree_MACD(1)),
		cree_ligne(SRC_PRIXS, MACD, 64, 64, 0, cree_MACD(1)),
		cree_ligne(SRC_PRIXS, MACD, 64, 64, 8, cree_MACD(1)),
		cree_ligne(SRC_PRIXS, MACD, 128, 8.0, 0, cree_MACD(1)),
		cree_ligne(SRC_PRIXS, MACD, 128, 8.0, 8, cree_MACD(1)),
		cree_ligne(SRC_PRIXS, MACD, 128, 128, 0, cree_MACD(1)),
		cree_ligne(SRC_PRIXS, MACD, 128, 128, 8, cree_MACD(1)),
		cree_ligne(SRC_HIGH, MACD, 1, 1, 0, cree_MACD(1)),
		cree_ligne(SRC_HIGH, MACD, 1, 1, 8, cree_MACD(1)),
		cree_ligne(SRC_HIGH, MACD, 4, 4, 0, cree_MACD(1)),
		cree_ligne(SRC_HIGH, MACD, 4, 4, 8, cree_MACD(1)),
		cree_ligne(SRC_HIGH, MACD, 16, 1.0, 0, cree_MACD(1)),
		cree_ligne(SRC_HIGH, MACD, 16, 1.0, 8, cree_MACD(1)),
		cree_ligne(SRC_HIGH, MACD, 16, 16, 0, cree_MACD(1)),
		cree_ligne(SRC_HIGH, MACD, 16, 16, 8, cree_MACD(1)),
		cree_ligne(SRC_HIGH, MACD, 64, 4.0, 0, cree_MACD(1)),
		cree_ligne(SRC_HIGH, MACD, 64, 4.0, 8, cree_MACD(1)),
		cree_ligne(SRC_HIGH, MACD, 64, 64, 0, cree_MACD(1)),
		cree_ligne(SRC_HIGH, MACD, 64, 64, 8, cree_MACD(1)),
		cree_ligne(SRC_HIGH, MACD, 128, 8.0, 0, cree_MACD(1)),
		cree_ligne(SRC_HIGH, MACD, 128, 8.0, 8, cree_MACD(1)),
		cree_ligne(SRC_HIGH, MACD, 128, 128, 0, cree_MACD(1)),
		cree_ligne(SRC_HIGH, MACD, 128, 128, 8, cree_MACD(1)),
		cree_ligne(SRC_LOW, MACD, 1, 1, 0, cree_MACD(1)),
		cree_ligne(SRC_LOW, MACD, 1, 1, 8, cree_MACD(1)),
		cree_ligne(SRC_LOW, MACD, 4, 4, 0, cree_MACD(1)),
		cree_ligne(SRC_LOW, MACD, 4, 4, 8, cree_MACD(1)),
		cree_ligne(SRC_LOW, MACD, 16, 1.0, 0, cree_MACD(1)),
		cree_ligne(SRC_LOW, MACD, 16, 1.0, 8, cree_MACD(1)),
		cree_ligne(SRC_LOW, MACD, 16, 16, 0, cree_MACD(1)),
		cree_ligne(SRC_LOW, MACD, 16, 16, 8, cree_MACD(1)),
		cree_ligne(SRC_LOW, MACD, 64, 4.0, 0, cree_MACD(1)),
		cree_ligne(SRC_LOW, MACD, 64, 4.0, 8, cree_MACD(1)),
		cree_ligne(SRC_LOW, MACD, 64, 64, 0, cree_MACD(1)),
		cree_ligne(SRC_LOW, MACD, 64, 64, 8, cree_MACD(1)),
		cree_ligne(SRC_LOW, MACD, 128, 8.0, 0, cree_MACD(1)),
		cree_ligne(SRC_LOW, MACD, 128, 8.0, 8, cree_MACD(1)),
		cree_ligne(SRC_LOW, MACD, 128, 128, 0, cree_MACD(1)),
		cree_ligne(SRC_LOW, MACD, 128, 128, 8, cree_MACD(1)),
		cree_ligne(SRC_VOLUMES, MACD, 1, 1, 0, cree_MACD(1)),
		cree_ligne(SRC_VOLUMES, MACD, 1, 1, 8, cree_MACD(1)),
		cree_ligne(SRC_VOLUMES, MACD, 4, 4, 0, cree_MACD(1)),
		cree_ligne(SRC_VOLUMES, MACD, 4, 4, 8, cree_MACD(1)),
		cree_ligne(SRC_VOLUMES, MACD, 16, 1.0, 0, cree_MACD(1)),
		cree_ligne(SRC_VOLUMES, MACD, 16, 1.0, 8, cree_MACD(1)),
		cree_ligne(SRC_VOLUMES, MACD, 16, 16, 0, cree_MACD(1)),
		cree_ligne(SRC_VOLUMES, MACD, 16, 16, 8, cree_MACD(1)),
		cree_ligne(SRC_VOLUMES, MACD, 64, 4.0, 0, cree_MACD(1)),
		cree_ligne(SRC_VOLUMES, MACD, 64, 4.0, 8, cree_MACD(1)),
		cree_ligne(SRC_VOLUMES, MACD, 64, 64, 0, cree_MACD(1)),
		cree_ligne(SRC_VOLUMES, MACD, 64, 64, 8, cree_MACD(1)),
		cree_ligne(SRC_VOLUMES, MACD, 128, 8.0, 0, cree_MACD(1)),
		cree_ligne(SRC_VOLUMES, MACD, 128, 8.0, 8, cree_MACD(1)),
		cree_ligne(SRC_VOLUMES, MACD, 128, 128, 0, cree_MACD(1)),
		cree_ligne(SRC_VOLUMES, MACD, 128, 128, 8, cree_MACD(1)),
	// ----
		cree_ligne(SRC_HIGH, CHIFFRE, 1, 1, 0, cree_CHIFFRE(1000)),
		cree_ligne(SRC_HIGH, CHIFFRE, 1, 1, 8, cree_CHIFFRE(1000)),
		cree_ligne(SRC_HIGH, CHIFFRE, 8, 8, 0, cree_CHIFFRE(1000)),
		cree_ligne(SRC_HIGH, CHIFFRE, 8, 8, 8, cree_CHIFFRE(1000)),
		cree_ligne(SRC_HIGH, CHIFFRE, 32, 32, 0, cree_CHIFFRE(1000)),
		cree_ligne(SRC_HIGH, CHIFFRE, 32, 32, 8, cree_CHIFFRE(1000)),
		cree_ligne(SRC_HIGH, CHIFFRE, 128, 128, 0, cree_CHIFFRE(1000)),
		cree_ligne(SRC_HIGH, CHIFFRE, 128, 128, 8, cree_CHIFFRE(1000)),
		cree_ligne(SRC_HIGH, CHIFFRE, 1, 1, 0, cree_CHIFFRE(10000)),
		cree_ligne(SRC_HIGH, CHIFFRE, 1, 1, 8, cree_CHIFFRE(10000)),
		cree_ligne(SRC_HIGH, CHIFFRE, 8, 8, 0, cree_CHIFFRE(10000)),
		cree_ligne(SRC_HIGH, CHIFFRE, 8, 8, 8, cree_CHIFFRE(10000)),
		cree_ligne(SRC_HIGH, CHIFFRE, 32, 32, 0, cree_CHIFFRE(10000)),
		cree_ligne(SRC_HIGH, CHIFFRE, 32, 32, 8, cree_CHIFFRE(10000)),
		cree_ligne(SRC_HIGH, CHIFFRE, 128, 128, 0, cree_CHIFFRE(10000)),
		cree_ligne(SRC_HIGH, CHIFFRE, 128, 128, 8, cree_CHIFFRE(10000)),
		cree_ligne(SRC_LOW, CHIFFRE, 1, 1, 0, cree_CHIFFRE(1000)),
		cree_ligne(SRC_LOW, CHIFFRE, 1, 1, 8, cree_CHIFFRE(1000)),
		cree_ligne(SRC_LOW, CHIFFRE, 8, 8, 0, cree_CHIFFRE(1000)),
		cree_ligne(SRC_LOW, CHIFFRE, 8, 8, 8, cree_CHIFFRE(1000)),
		cree_ligne(SRC_LOW, CHIFFRE, 32, 32, 0, cree_CHIFFRE(1000)),
		cree_ligne(SRC_LOW, CHIFFRE, 32, 32, 8, cree_CHIFFRE(1000)),
		cree_ligne(SRC_LOW, CHIFFRE, 128, 128, 0, cree_CHIFFRE(1000)),
		cree_ligne(SRC_LOW, CHIFFRE, 128, 128, 8, cree_CHIFFRE(1000)),
		cree_ligne(SRC_LOW, CHIFFRE, 1, 1, 0, cree_CHIFFRE(10000)),
		cree_ligne(SRC_LOW, CHIFFRE, 1, 1, 8, cree_CHIFFRE(10000)),
		cree_ligne(SRC_LOW, CHIFFRE, 8, 8, 0, cree_CHIFFRE(10000)),
		cree_ligne(SRC_LOW, CHIFFRE, 8, 8, 8, cree_CHIFFRE(10000)),
		cree_ligne(SRC_LOW, CHIFFRE, 32, 32, 0, cree_CHIFFRE(10000)),
		cree_ligne(SRC_LOW, CHIFFRE, 32, 32, 8, cree_CHIFFRE(10000)),
		cree_ligne(SRC_LOW, CHIFFRE, 128, 128, 0, cree_CHIFFRE(10000)),
		cree_ligne(SRC_LOW, CHIFFRE, 128, 128, 8, cree_CHIFFRE(10000))
	};
	//
	Mdl_t * mdl = cree_mdl(GRAND_T, Y, insts, bloque);*/

	/*Mdl_t * mdl = ouvrire_mdl(GRAND_T, "mdl.bin");

	enregistrer_les_lignes_brute(mdl, "lignes_brute.bin");

	plumer_mdl(mdl);

	//	================= Initialisation ==============
	uint t0 = DEPART;
	uint t1 = t0 + ROND_MODULO((FIN-DEPART), (16*16));
	printf("t0=%i t1=%i FIN=%i (t1-t0=%i, %%(16*16)=%i)\n", t0, t1, FIN, t1-t0, (t1-t0)%(16*16));
	//
	plume_pred(mdl, t0, t1);
	//
	srand(time(NULL));
#define PERTURBATIONS 0
	//
	uint REP = 300;
	FOR(0, rep, REP) {
		perturber(mdl, 50);
		perturber_filtres(mdl, 50);
		optimisation_mini_packet(
			mdl,
			t0, t1, GRAND_T,
			alpha, 1.0,
			RMSPROP, 40,
			pourcent_masque,
			//pourcent_masque_nulle,
			PERTURBATIONS,
			optimiser_tous_les);
		mdl_gpu_vers_cpu(mdl);
		ecrire_mdl(mdl, "mdl.bin");
		plume_pred(mdl, t0, t1);
		//
		printf("===================================================\n");
		printf("==================TERMINE %i/%i=======================\n", rep+1, REP);
		printf("===================================================\n");
	}
	//
	mdl_gpu_vers_cpu(mdl);
	ecrire_mdl(mdl, "mdl.bin");
	liberer_mdl(mdl);*/

	//	-- Fin --
	liberer_tout();
};