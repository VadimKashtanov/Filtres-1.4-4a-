#include "hip/hip_runtime.h"
#include "mdl.cuh"

#include "filtres_prixs.cuh"
#include "dot1d.cuh"
#include "lstm1d.cuh"
#include "dot1d_blk.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

mdl_inst_f cree_inst[INSTS] = {
	cree_filtre_prixs,
	cree_dot1d,
	cree_dot1d_blk
};

mdl_f_f inst_f [INSTS] = {
	f_filtres_prixs,
	f_dot1d,
	f_dot1d_blk
};

mdl_f_f inst_df[INSTS] = {
	df_filtres_prixs,
	df_dot1d,
	df_dot1d_blk
};

char * nom_inst[INSTS] = {
	"filtres_prixs",
	"dot1d        ",
	"dot1d bloques"
};

mdl_inst_f plume_inst[INSTS] = {
	plume_filtre_prixs,
	plume_dot1d,
	plume_dot1d_blk
};

static void calculer_normalisee__et__dif_normalisee(Mdl_t * mdl) { 
	FOR(0, b, BLOQUES) {
		FOR(DEPART-MAX_DECALES, t, FIN) {
			//	_max & _min pour ce filtre-8
			float _max, _min;
			if (mdl->type_de_norme[b] == NORME_CLASSIQUE) {
				_max = mdl->bloque[b]->brute[t - 0*mdl->bloque[b]->intervalle];
				_min = mdl->bloque[b]->brute[t - 0*mdl->bloque[b]->intervalle];
				FOR(1, i, N_FLTR) {
					if (_max < mdl->bloque[b]->brute[t-i*mdl->bloque[b]->intervalle])
						_max = mdl->bloque[b]->brute[t-i*mdl->bloque[b]->intervalle];
					if (_min > mdl->bloque[b]->brute[t-i*mdl->bloque[b]->intervalle])
						_min = mdl->bloque[b]->brute[t-i*mdl->bloque[b]->intervalle];
				}
			} else if (mdl->type_de_norme[b] == NORME_THEORIQUE) {
				_min = mdl->min_theorique[b];
				_max = mdl->max_theorique[b];
			} else {
				ERR("Norme ni classique, ni theorique, mdl->type_de_norme[b] == %i", mdl->type_de_norme[b]);
			};

			//
			FOR(0, i, N_FLTR) {
				mdl->normalisee[b*PRIXS*N_FLTR+t*N_FLTR+i] = ( mdl->bloque[b]->brute[t-i*mdl->bloque[b]->intervalle] - _min)/( _max - _min );
				//if (mdl->normalisee[b*PRIXS*N_FLTR+t*N_FLTR+i]==0) printf("%i %i %i\n", b,t,i);
			}
		};

		FOR(DEPART-MAX_DECALES, t, FIN) {
			FOR(1, i, N_FLTR)
				mdl->dif_normalisee[b*PRIXS*N_FLTR+t*N_FLTR+i] = mdl->normalisee[b*PRIXS*N_FLTR+t*N_FLTR+i] - mdl->normalisee[b*PRIXS*N_FLTR+t*N_FLTR+i-1];
			mdl->dif_normalisee[b*PRIXS*N_FLTR+t*N_FLTR+N_FLTR+0] = 0.f;
		}
	}

	mdl->normalisee__d     = cpu_vers_gpu<float>(mdl->normalisee,     BLOQUES * PRIXS * N_FLTR);
	mdl->dif_normalisee__d = cpu_vers_gpu<float>(mdl->dif_normalisee, BLOQUES * PRIXS * N_FLTR);
};

static uint tout_zeroiser[C] = UNIFORME_C(1);

Mdl_t * cree_mdl(
	uint T,
	uint Y[C],
	uint insts[C],
	ema_int_t * bloque[BLOQUES]
) {
	ASSERT(Y[C-1] == P);
	ASSERT(Y[ 0 ] == BLOQUES * F_PAR_BLOQUES);
	ASSERT(insts[C-1] == DOT1D);				//	Afin d'assurer un Y=inst_VARS
	
	Mdl_t * mdl = alloc<Mdl_t>(1);

	mdl->T = T;

	//
	FOR(0, i, BLOQUES) {
		mdl->bloque[i]  = bloque[i];
		mdl->decales[i] = bloque[i]->decale;
		mdl->intervalles[i] = bloque[i]->intervalle;
		//
		mdl->type_de_norme[i] = bloque[i]->type_de_norme;
		mdl->min_theorique[i] = bloque[i]->min_theorique;
		mdl->max_theorique[i] = bloque[i]->max_theorique;
	};

	mdl->decales__d = cpu_vers_gpu<uint>(mdl->decales, BLOQUES);
	mdl->intervalles__d = cpu_vers_gpu<uint>(mdl->intervalles, BLOQUES);

	//
	calculer_normalisee__et__dif_normalisee(mdl);
	//raise(SIGINT);

	//	Architecture
	memcpy(mdl->insts, insts, sizeof(uint) * C);
	memcpy(mdl->Y,         Y, sizeof(uint) * C);

	//	Allocation
	mdl->total_POIDS = 0;
	FOR(0, c, C) {
		if (c>0) ASSERT(insts[c] != 0);
		ASSERT(Y[c] <= MAX_Y);
		//
		cree_inst[insts[c]](mdl, c);
		//
		//mdl->p [c] = alloc<float>(mdl->inst_POIDS[c]);
		mdl->y [c] = alloc<float>(mdl->inst_VARS [c] * T);
		mdl->l [c] = alloc<float>(mdl->inst_LOCDS[c] * T);
		mdl->dy[c] = alloc<float>(mdl->inst_VARS [c] * T);
		mdl->dp[c] = alloc<float>(mdl->inst_POIDS[c]);
		//
		mdl->p__d [c] = cpu_vers_gpu<float>(mdl->p[c], mdl->inst_POIDS[c]);
		mdl->y__d [c] = cudalloc<float>(mdl->inst_VARS [c] * T);
		mdl->l__d [c] = cudalloc<float>(mdl->inst_LOCDS[c] * T);
		mdl->dy__d[c] = cudalloc<float>(mdl->inst_VARS [c] * T);
		mdl->dp__d[c] = cudalloc<float>(mdl->inst_POIDS[c]);
		//
		mdl->total_POIDS += mdl->inst_POIDS[c];
	}
	ASSERT(mdl->inst_DEPART_SORTIE[C-1] == 0);
	//
	mdl_normer_les_filtres(mdl);
	//
	mdl_zero_deriv_gpu(mdl, tout_zeroiser);
	mdl_zero_deriv_cpu(mdl, tout_zeroiser);
	//
	return mdl;
};

void mdl_normer_les_filtres(Mdl_t * mdl) {
	FOR(0, b, BLOQUES) {
		FOR(0, f, F_PAR_BLOQUES) {
			float min, max;
			if (mdl->type_de_norme[b] == NORME_CLASSIQUE) {
				max=mdl->p[0][b*F_PAR_BLOQUES*N + f*N+0];
				min=mdl->p[0][b*F_PAR_BLOQUES*N + f*N+0];
				FOR(1, i, N) {
					if (max < mdl->p[0][b*F_PAR_BLOQUES*N + f*N+i]) max = mdl->p[0][b*F_PAR_BLOQUES*N + f*N+i];
					if (min > mdl->p[0][b*F_PAR_BLOQUES*N + f*N+i]) min = mdl->p[0][b*F_PAR_BLOQUES*N + f*N+i];
				}
			} else if (mdl->type_de_norme[b] == NORME_THEORIQUE) {
				min = 0.0;	//se sont des filtres
				max = 1.0;	//pas les natures
			} else {
				ERR("mdl->type_de_norme[b]=%i", mdl->type_de_norme[b]);
			}
			//
			FOR(0, i, N) mdl->p[0][b*F_PAR_BLOQUES*N + f*N+i] = (mdl->p[0][b*F_PAR_BLOQUES*N + f*N+i]-min)/(max-min);
		};
	}
	CONTROLE_CUDA(hipMemcpy(mdl->p__d[0], mdl->p[0], sizeof(float)*BLOQUES*F_PAR_BLOQUES*N, hipMemcpyHostToDevice))
};

PAS_OPTIMISER()
void mdl_verif(Mdl_t * mdl) {
	FOR(1, c, C) {
		float * r = gpu_vers_cpu<float>(mdl->p__d[c], mdl->inst_POIDS[c]);
		FOR(0, i, mdl->inst_POIDS[c]) ASSERT(fabs(r[i]-mdl->p[c][i]) < 0.01);
		free(r);
	}
};

PAS_OPTIMISER()
void mdl_gpu_vers_cpu(Mdl_t * mdl) {
	FOR(0, c, C) {
		CONTROLE_CUDA(hipMemcpy(mdl->p[c],  mdl->p__d[c],  sizeof(float)*mdl->inst_POIDS[c],       hipMemcpyDeviceToHost));
		CONTROLE_CUDA(hipMemcpy(mdl->y[c],  mdl->y__d[c],  sizeof(float)*mdl->inst_VARS[c]*mdl->T,  hipMemcpyDeviceToHost));
		CONTROLE_CUDA(hipMemcpy(mdl->l[c],  mdl->l__d[c],  sizeof(float)*mdl->inst_LOCDS[c]*mdl->T, hipMemcpyDeviceToHost));
		CONTROLE_CUDA(hipMemcpy(mdl->dy[c], mdl->dy__d[c], sizeof(float)*mdl->inst_VARS[c]*mdl->T,  hipMemcpyDeviceToHost));
		CONTROLE_CUDA(hipMemcpy(mdl->dp[c], mdl->dp__d[c], sizeof(float)*mdl->inst_POIDS[c],       hipMemcpyDeviceToHost));
	}
}

PAS_OPTIMISER()
void mdl_cpu_vers_gpu(Mdl_t * mdl) {
	FOR(0, c, C) {
		CONTROLE_CUDA(hipMemcpy(mdl->p__d[c],  mdl->p[c],  sizeof(float)*mdl->inst_POIDS[c],       hipMemcpyHostToDevice));
		CONTROLE_CUDA(hipMemcpy(mdl->y__d[c],  mdl->y[c],  sizeof(float)*mdl->inst_VARS[c]*mdl->T,  hipMemcpyHostToDevice));
		CONTROLE_CUDA(hipMemcpy(mdl->l__d[c],  mdl->l[c],  sizeof(float)*mdl->inst_LOCDS[c]*mdl->T, hipMemcpyHostToDevice));
		CONTROLE_CUDA(hipMemcpy(mdl->dy__d[c], mdl->dy[c], sizeof(float)*mdl->inst_VARS[c]*mdl->T,  hipMemcpyHostToDevice));
		CONTROLE_CUDA(hipMemcpy(mdl->dp__d[c], mdl->dp[c], sizeof(float)*mdl->inst_POIDS[c],       hipMemcpyHostToDevice));
	}
};

PAS_OPTIMISER()
void liberer_mdl(Mdl_t * mdl) {
	CONTROLE_CUDA(hipFree(mdl->decales__d));
	FOR(0, c, C) {
		free(mdl->p [c]);
		free(mdl->y [c]);
		free(mdl->l [c]);
		free(mdl->dy[c]);
		free(mdl->dp[c]);
		//
		CONTROLE_CUDA(hipFree(mdl->p__d [c]));
		CONTROLE_CUDA(hipFree(mdl->y__d [c]));
		CONTROLE_CUDA(hipFree(mdl->l__d [c]));
		CONTROLE_CUDA(hipFree(mdl->dy__d[c]));
		CONTROLE_CUDA(hipFree(mdl->dp__d[c]));
	}
};

PAS_OPTIMISER()
void mdl_zero_cpu(Mdl_t * mdl) {
	FOR(0, c, C) {
		memset(mdl->y [c], 0, sizeof(float) * mdl->inst_VARS [c] * mdl->T);
	}
};

PAS_OPTIMISER()
void mdl_zero_gpu(Mdl_t * mdl) {
	FOR(0, c, C) {
		CONTROLE_CUDA(hipMemset(mdl->y__d [c], 0, sizeof(float) * mdl->inst_VARS [c] * mdl->T));
	}
};

PAS_OPTIMISER()
void mdl_zero_deriv_cpu(Mdl_t * mdl, uint zeroiser[C]) {
	FOR(0, c, C) {
		if (zeroiser[c]) {
			memset(mdl->dy[c], 0, sizeof(float) * mdl->inst_VARS [c] * mdl->T);
			memset(mdl->dp[c], 0, sizeof(float) * mdl->inst_POIDS[c]);
		}
	}
};

PAS_OPTIMISER()
void mdl_zero_deriv_gpu(Mdl_t * mdl, uint zeroiser[C]) {
	FOR(0, c, C) {
		if (zeroiser[c]) {
			CONTROLE_CUDA(hipMemset(mdl->dy__d[c], 0, sizeof(float) * mdl->inst_VARS [c] * mdl->T));
			CONTROLE_CUDA(hipMemset(mdl->dp__d[c], 0, sizeof(float) * mdl->inst_POIDS[c]));
		}
	}
};