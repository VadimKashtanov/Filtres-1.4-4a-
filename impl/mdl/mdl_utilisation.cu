#include "hip/hip_runtime.h"
#include "mdl.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

#define COEF_POTENTIEL 2

float mdl_les_gains(Mdl_t * mdl, uint t0, uint t1, uint mode) {
	ASSERT(mdl->T == (t1-t0));
	float * _y = gpu_vers_cpu<float>(mdl->y__d[C-1], PRIXS*P);
	float somme = 0;
	float potentiel = 0;
	FOR(t0, t, t1) {
		somme     += powf(fabs(prixs[t+1]/prixs[t]-1),COEF_POTENTIEL) * (signe((prixs[t+1]/prixs[t]-1)) == signe(_y[(t-t0)*P+0]));
		potentiel += powf(fabs(prixs[t+1]/prixs[t]-1),COEF_POTENTIEL);
	}
	return somme / potentiel;
};

float mdl_score(Mdl_t * mdl, uint t0, uint t1, uint mode) {
	ASSERT(mdl->T == (t1-t0));
	if (mode == 0) mdl_zero_cpu(mdl);
	else           mdl_zero_gpu(mdl);
	//
	mdl_f(mdl, t0, t1, mode);
	//
	float somme_score;
	if (mode == 0) somme_score =  intel_somme_score(mdl->y[C-1],    t0, (t1-t0));
	else           somme_score = nvidia_somme_score(mdl->y__d[C-1], t0, (t1-t0));
	//
	if (mode == 0) return  intel_score_finale(somme_score, (t1-t0));
	else           return nvidia_score_finale(somme_score, (t1-t0));
};

float* mdl_pred(Mdl_t * mdl, uint t0, uint t1, uint mode) {
	ASSERT(mdl->T == (t1-t0));
	if (mode == 0) mdl_zero_cpu(mdl);
	else           mdl_zero_gpu(mdl);
	//
	mdl_f(mdl, t0, t1, mode);
	if (mode == 0) return  intel_prediction(mdl->y[C-1], t0, (t1-t0));
	else           return nvidia_prediction(mdl->y__d[C-1], t0, (t1-t0));
};

void mdl_aller_retour(Mdl_t * mdl, uint t0, uint t1, uint mode) {
	ASSERT(mdl->T == (t1-t0));
	if (mode == 0) mdl_zero_cpu(mdl);
	else           mdl_zero_gpu(mdl);
	mdl_f(mdl, t0, t1, mode);
	//
	float somme_score;
	if (mode == 0) somme_score =  intel_somme_score(mdl->y[C-1], t0, (t1-t0));
	else           somme_score = nvidia_somme_score(mdl->y__d[C-1], t0, (t1-t0));
	//
	float d_score;
	if (mode == 0) d_score =  d_intel_score_finale(somme_score, (t1-t0));
	else           d_score = d_nvidia_score_finale(somme_score, (t1-t0));
	//
	if (mode == 0)  d_intel_somme_score(d_score, mdl->y[C-1],    mdl->dy[C-1], t0, (t1-t0));
	else           d_nvidia_somme_score(d_score, mdl->y__d[C-1], mdl->dy__d[C-1], t0, (t1-t0));
	mdl_df(mdl, t0, t1, mode);
};