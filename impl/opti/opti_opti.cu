#include "hip/hip_runtime.h"
#include "opti.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

static uint tout_zeroiser[C] = UNIFORME_C(1);

PAS_OPTIMISER()
void __interne_optimiser(
	Mdl_t * mdl,
	uint t0, uint t1,
	float * alpha, float div,
	uint methode, uint I,
	uint ** masque,
	uint PERTURBATIONS,
	uint zero_accumulation_tous_les[C])
{
	mdl_zero_deriv_gpu(mdl, tout_zeroiser);
	//
	//	Cree les listes pour les `hist` si un opti en a besoin 
	Opti_classe_t opti_classe;
	if      (methode == SGD)     opti_classe.sgd     = (uint)NULL;
	else if (methode == RMSPROP) opti_classe.rmsprop = cree_rmsprop(mdl);
	else if (methode == ADAM)    opti_classe.adam    = cree_adam(mdl);
	else ERR("Pas de methode %i d'optimisation", methode);
	
	//	Plumer grad pour mieux y voire
	mdl_plume_grad(mdl, t0, t1);
	
	/* ------- Optimisation ----------- */
	uint zeroiser[C];
	FOR(0, i, I) {
		//
		FOR(0, j, C) {
			if (i % zero_accumulation_tous_les[j] == 0)
				zeroiser[j] = 1;
			else
				zeroiser[j] = 0;
		}
		//
		perturber(mdl, PERTURBATIONS);
		mdl_aller_retour(mdl, t0, t1, 3);
		
		//	--------- * Optimisation * -------------
#define optimiser_la_couche zeroiser
		if (methode == SGD)     opti_simple (zero_accumulation_tous_les, optimiser_la_couche, mdl, alpha, div, masque);
		if (methode == RMSPROP) opti_rmsprop(zero_accumulation_tous_les, optimiser_la_couche, mdl, opti_classe.rmsprop, alpha, div, masque);
		if (methode == ADAM)    opti_adam   (zero_accumulation_tous_les, optimiser_la_couche, mdl, opti_classe.adam,    alpha, div, masque);
		//
		mdl_zero_deriv_gpu(mdl, zeroiser);
		//
		mdl_normer_les_filtres(mdl);
		//
		if (i % 5 == 0) {
			float* __pred = mdl_pred(mdl, t0, t1, 3);
			float  _score = mdl_score(mdl, t0, t1, 3);
			//
			float les_gains = mdl_les_gains(mdl, t0, t1, 3);
			//
			printf("%3.i/%3.i| perf={", i, I);
			FOR(0, p, P) printf("%+f%%, ", 100*__pred[p]);
			free(__pred);
			printf("} score=\033[93m%+f\033[0m (%%.potentiel=%+f)\n", _score, les_gains);
			if (fabs(_score) < 0.00001) {
				printf("Score < 0.00001 => Fin d'optimisation\n");
				break;
			}
		}
	}

	//	Liberer
	if      (methode == SGD)     opti_classe.sgd = 0;
	else if (methode == RMSPROP) liberer_rmsprop(opti_classe.rmsprop);
	else if (methode == ADAM)    liberer_adam   (opti_classe.adam   );
};

void optimiser(
	Mdl_t * mdl,
	uint t0, uint t1,
	float * alpha, float div,
	uint methode, uint I,
	float * pourcent_masque,
	uint PERTURBATIONS,
	uint zero_accumulation_tous_les[C])
{
	Masque_t * masque = cree_masque(mdl, pourcent_masque);
	//
	__interne_optimiser(
		mdl,
		t0, t1,
		alpha, div,
		methode, I,
		masque->masque,
		PERTURBATIONS,
		zero_accumulation_tous_les);
	//
	sortire_masque(mdl, masque);
};