#include "hip/hip_runtime.h"
#include "main.cuh"

#include "../impl_tmpl/tmpl_etc.cu"

static float filtre(uint depart, float * x, float * f, uint intervalle, uint decale, uint type_norme, float _min, float _max) {
	float normer_x[N];
	//
	FOR(0, i, N) normer_x[i] = x[depart - (decale+i)*intervalle];
	//
	if (type_norme == NORME_CLASSIQUE) {
		_min=normer_x[0];
		_max=normer_x[0];
		//
		FOR(1, i, N) {
			float a = normer_x[i];
			if (a > _max) _max = a;
			if (a < _min) _min = a;
		}
	} else if (type_norme == NORME_THEORIQUE) {
		// rien
	} else {
		ERR("type_norme == %i", type_norme);
	}
	//
	FOR(0, i, N) normer_x[i] = (normer_x[i]-_min)/(_max-_min);
	//
	float s = 0, d = 0;
	float f_nouveau = f[0];
	float x_nouveau = normer_x[0];
	s += sqrtf(1 + fabs(x_nouveau - f_nouveau));
	float f_avant = f_nouveau;
	float x_avant = x_nouveau;
	FOR(1, i, N) {
		f_nouveau = f[i];
		x_nouveau = normer_x[i];
		s += sqrtf(1 + fabs(  x_nouveau   -   f_nouveau  ));
		d += powf((1 + fabs((x_nouveau-x_avant) - (f_nouveau-f_avant))), 2);
		f_avant   = f_nouveau;
		x_avant   = x_nouveau;
	};

	s = s/8-1;
	d = d/7-1;

	return 2*expf(-s*s -d*d)-1;
};


int main(int argc, char ** argv) {
	srand(0);
	hipSetDevice(0);
	//
	FILE * fp = fopen(argv[1], "rb");
	//
	uint Y[C];
	FREAD(Y, sizeof(uint), C, fp);
	uint insts[C];
	FREAD(insts, sizeof(uint), C, fp);
	//
	//
	//
	//
	uint PRIXS_bitget;
	FREAD(&PRIXS_bitget, sizeof(uint), 1, fp);
	uint intervalles[BLOQUES], decales[BLOQUES];
	FREAD(intervalles, sizeof(uint), BLOQUES, fp);
	FREAD(decales,     sizeof(uint), BLOQUES, fp);
	//
	//
	//
	uint type_norme[BLOQUES];
	float _min[BLOQUES], _max[BLOQUES];
	FREAD(type_norme, sizeof(uint), BLOQUES, fp);
	FREAD(_min,       sizeof(float), BLOQUES, fp);
	FREAD(_max,       sizeof(float), BLOQUES, fp);
	//
	//
	//
	float * lignes = alloc<float>(PRIXS_bitget*BLOQUES);
	FREAD(lignes, sizeof(float), PRIXS_bitget*BLOQUES, fp);
	//
	float * poids[C];
	FOR(0, c, C) {
		uint POIDS;
		FREAD(&POIDS, sizeof(uint), 1, fp);
		poids[c] = alloc<float>(POIDS);
		FREAD(poids[c], sizeof(float), POIDS, fp);
	}
	//
	fclose(fp);

	//	------------- Calcule ----------------
	float * y_avant   = alloc<float>( PRIXS_bitget*MAX_Y );
	float * y_nouveau = alloc<float>( PRIXS_bitget*MAX_Y );
	//
	FOR(0, f, BLOQUES*F_PAR_BLOQUES) {
		uint b = (f - (f % F_PAR_BLOQUES)) / F_PAR_BLOQUES;
		FOR(DEPART, t, PRIXS_bitget) {
			y_nouveau[t*MAX_Y + f] = filtre(
				b*PRIXS_bitget + t,
				lignes,
				poids[0] + f*N,
				intervalles[b], decales[b],
				type_norme[b],
				_min[b], _max[b]
			);
		}
	};
	FOR(0, i, PRIXS_bitget*MAX_Y) y_avant[i] = y_nouveau[i];
	//
	FOR(1, c, C) {
		if (insts[c] == DOT1D) {
			uint X = Y[c-1];
			FOR(0, i, Y[c]) {
				FOR(DEPART, t, PRIXS_bitget) {
					float s = poids[c][(X+1)*i + X-1+1];
					FOR(0, j, X) s += poids[c][(X+1)*i + j] * y_avant[t*MAX_Y + j];
					y_nouveau[t*MAX_Y + i] = tanh(s);
				};
			};
		} else if (insts[c] == DOT1D_BLK) {
#include "dot1d_blk.cuh"
			uint  X = Y[c-1];
			uint _Y = Y[ c ];
			//
			uint X_blk =  X / DOT1D_BLK_BLOQUES;
			uint Y_blk = _Y / DOT1D_BLK_BLOQUES;
			uint P_blk =  ( X_blk + 1 ) * Y_blk;
			//
			FOR(DEPART, t, PRIXS_bitget) {
				FOR(0, blk, DOT1D_BLK_BLOQUES) {
					//
					uint depart_y = blk * Y_blk;
					uint depart_x = blk * X_blk;
					uint depart_p = blk * P_blk;
					//
					FOR(0, y, Y_blk) {
						float s = poids[c][depart_p + (X_blk+1)*y + (X_blk+1)+1];
						FOR(0, j, X_blk)
							s += poids[c][depart_p + (X_blk+1)*y] * y_avant[t*MAX_Y + depart_x + j];
						y_nouveau[t*MAX_Y + depart_y + y] = tanh(s);
					};
				};
			}
		} else {
			ERR("Inst = %i", insts[c]);
		}

		/*#pragma omp parallel
		#pragma omp for*/
		FOR(0, i, PRIXS_bitget*MAX_Y) y_avant[i] = y_nouveau[i];
	};

	//	---------- Ecrire Resultat ----------
	fp = fopen(argv[1], "wb");
	//
	float res[PRIXS_bitget];
	FOR(DEPART, t, PRIXS_bitget) res[t] = y_nouveau[t*MAX_Y + 0];
	FWRITE(res+DEPART, sizeof(float), (PRIXS_bitget-DEPART), fp);
	//
	fclose(fp);
}